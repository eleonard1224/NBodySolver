#include "hip/hip_runtime.h"
// Implementation of nbody cuda calls
#include "nbody.h"

__global__ void calculate_positions(void *devX, void *devV, void *devA, void *posR, int N, int p, int nt, float dt) {

    float4 *globalR = (float4 *) posR;
    int it;
    for(it = 1; it < nt; it++) {
        calculate_forces(devX, devV, devA, globalR, N, p, it, dt);
    }
}

__device__ void calculate_forces(void *devX, void *devV, void *devA, float4 *globalR, int N, int p, int it, float dt) { 

    extern __shared__ float4 shPosition[];    
    float4 *globalX = (float4 *)devX;  
    float4 *globalV = (float4 *)devV;    
    float4 *globalA = (float4 *)devA;   
    float4 myPosition;   
    int i, tile;   
    float3 acc = {0.0f, 0.0f, 0.0f};   
    int gtid = blockIdx.x * blockDim.x + threadIdx.x;   
    myPosition = globalX[gtid];
    for (i = 0, tile = 0; i < N; i += p, tile++) { 
        int idx = tile * blockDim.x + threadIdx.x; 
        shPosition[threadIdx.x] = globalX[idx];   
        __syncthreads();     
        acc = tile_calculation(myPosition, acc); 
        __syncthreads();   
    }
    float4 acc4 = {acc.x, acc.y, acc.z, 0.0f};   
    globalA[gtid] = acc4; 
    globalX[gtid] = globalX[gtid] + globalV[gtid]*dt + globalA[gtid]*dt*0.5f;
    globalV[gtid] += (acc4*dt); 
    globalR[N*it+gtid] = globalX[gtid];
} 

__device__ float3 tile_calculation(float4 myPosition, float3 accel) {   

    int i;   
    extern __shared__ float4 shPosition[];    
    for (i = 0; i < blockDim.x; i++) {     
        accel = bodyBodyInteraction(myPosition, shPosition[i], accel);   
    }
    return accel; 
} 

__device__ float3 bodyBodyInteraction(float4 bi, float4 bj, float3 ai) {   

    float3 r;   
    r.x = bj.x - bi.x;   r.y = bj.y - bi.y;   r.z = bj.z - bi.z; 
    float distSqr = r.x * r.x + r.y * r.y + r.z * r.z + EPS2;   
    float distSixth = distSqr * distSqr * distSqr;   
    float invDistCube = 1.0f/sqrtf(distSixth);   
    float s = bj.w * invDistCube;   
    ai.x += r.x * s;   ai.y += r.y * s;   ai.z += r.z * s;   
    return ai; 
} 

__device__ inline float4& operator +=(float4& a, const float4& b) {
    a.x += b.x;
    a.y += b.y;
    a.z += b.z;
    a.w += b.w;
    return a;
}

__device__ inline float4 operator +(const float4& a, const float4& b) {
    float4 c;
    c.x = a.x + b.x;
    c.y = a.y + b.y;
    c.z = a.z + b.z;
    c.w = a.w + b.w;
    return c;
}

__device__ inline float4 operator *(const float4& a, const float& b) {
    float4 c;
    c.x = b*a.x;
    c.y = b*a.y;
    c.z = b*a.z;
    c.w = b*a.w;
    return c;
}