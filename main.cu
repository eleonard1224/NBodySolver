#include "hip/hip_runtime.h"
// main file for nbody functions
#include <iostream>
#include <math.h>
#include "nbody.h"
#include "video.h"

int main(void) {

    int N = 2; // Number of particles
    float *devX, *devV, *devA; // Arrays which hold positions, velocities, and accelerations of particles
    float *posR; // posR has 4*N_particles*N_time_steps entries - stores the locations of the particles over all the timesteps
    int nt = 4; // Number of time steps
    float dt = 0.1f; // Time step size

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&devX, 4*N*sizeof(float));
    hipMallocManaged(&devV, 4*N*sizeof(float));
    hipMallocManaged(&devA, 4*N*sizeof(float));
    hipMallocManaged(&posR, 4*N*nt*sizeof(float));

    // Initialize devX, devV, and devA arrays with zero floats.
    int i, j;
    for (i = 0; i < 4*N; i++) {
        devX[i] = 0.0f;
        devV[i] = 0.0f;
        devA[i] = 0.0f;
    }
    // Set Initial Positions of Particles
    devX[0] = 1.0f; devX[3] = 1.0f; devX[4] = -1.0f; devX[7] = 1.0f;
    // Copy over initial positions of particles to posR
    // posR[0] = devX[0]; posR[1] = devX[1]; posR[2] = devX[2];
    // posR[4] = devX[4]; posR[5] = devX[5]; posR[6] = devX[6];
    for(i = 0; i < 4*N; i += 4) {
        for(j = 0; j < 3; j++) {
            posR[i+j] = devX[i+j];
        }
    }

    // Run the function on using the GPU.
    calculate_positions<<<2, 1, N*sizeof(float4)>>>(devX, devV, devA, posR, N, 1, nt, dt); 
    hipDeviceSynchronize();

    // // Print-outs for testing purposes
    // cout << "Accelerations" << endl;
    // for (int i = 0; i < 4*N; i++) {
    //     cout << devA[i] << " ";
    // }
    // cout << endl;

    // cout << "Positions" << endl;
    // for (int i = 0; i < 4*N*nt; i++) {
    //     cout << posR[i] << " ";
    // }
    // cout << endl;

    Mat img = create_image(posR, N, 0, 5.0f, 5.0f, 512, 512);
}