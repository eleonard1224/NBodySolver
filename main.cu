#include "hip/hip_runtime.h"
// main file for nbody functions
#include <iostream>
#include <math.h>
#include "nbody.h"
#include "video.h"

int main(void) {

    int N = 4; // Number of particles
    float *devX, *devV, *devA; // Arrays which hold positions, velocities, and accelerations of particles
    float *posR; // posR has 4*N_particles*N_time_steps entries - stores the locations of the particles over all the timesteps
    int nt = 160; // Number of time steps
    float dt = 0.1f; // Time step size

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&devX, 4*N*sizeof(float));
    hipMallocManaged(&devV, 4*N*sizeof(float));
    hipMallocManaged(&devA, 4*N*sizeof(float));
    hipMallocManaged(&posR, 4*N*nt*sizeof(float));

    // Initialize devX, devV, and devA arrays with zero floats.
    int i, j;
    for (i = 0; i < 4*N; i++) {
        devX[i] = 0.0f;
        devV[i] = 0.0f;
        devA[i] = 0.0f;
    }
    // Set Initial Positions of Particles
    float c1 = 5.0f;
    float c2 = -5.0f;
    devX[0] = c1; devX[1] = c1; devX[3] = 1.0f; // Particle # 1
    devX[4] = c2; devX[5] = c1; devX[7] = 1.0f; // Particle # 2
    devX[8] = c2; devX[9] = c2; devX[11] = 1.0f; // Particle # 3
    devX[12] = c1; devX[13] = c2; devX[15] = 1.0f; // Particle # 4
    // Copy over initial positions of particles to posR
    // posR[0] = devX[0]; posR[1] = devX[1]; posR[2] = devX[2];
    // posR[4] = devX[4]; posR[5] = devX[5]; posR[6] = devX[6];
    for(i = 0; i < 4*N; i += 4) {
        for(j = 0; j < 3; j++) {
            posR[i+j] = devX[i+j];
        }
    }

    // Run the function on using the GPU.
    calculate_positions<<<N, 2, N*sizeof(float4)>>>(devX, devV, devA, posR, N, 1, nt, dt); 
    hipDeviceSynchronize();

    // // Print-outs for testing purposes
    // cout << "Accelerations" << endl;
    // for (int i = 0; i < 4*N; i++) {
    //     cout << devA[i] << " ";
    // }
    // cout << endl;

    // cout << "Positions" << endl;
    // for (int i = 0; i < 4*N*nt; i++) {
    //     cout << posR[i] << " ";
    // }
    // cout << endl;

    create_video(posR, N, nt, 20.0f, 20.0f, 512, 512, "test.avi");
    // Mat img = create_image(posR, N, 0, 5.0f, 5.0f, 512, 512);
}